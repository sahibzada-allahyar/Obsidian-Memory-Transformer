#include "hip/hip_runtime.h"
#include "cuda_ops.cuh"
#include <cstdio>

namespace cuda_ops {

__global__ void addArrays(const float* a, const float* b, float* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

void vectorAdd(const float* hostA, const float* hostB, float* hostC, int size) {
    // Declare device pointers
    float *deviceA, *deviceB, *deviceC;
    
    // Allocate device memory
    hipMalloc(&deviceA, size * sizeof(float));
    hipMalloc(&deviceB, size * sizeof(float));
    hipMalloc(&deviceC, size * sizeof(float));
    
    // Copy inputs to device
    hipMemcpy(deviceA, hostA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, size * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addArrays<<<blocksPerGrid, threadsPerBlock>>>(deviceA, deviceB, deviceC, size);
    
    // Copy result back to host
    hipMemcpy(hostC, deviceC, size * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    
    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}
